#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <thread>

using namespace std;
using namespace std::chrono;
// CSR matrix class
class matCsr {
public:
    vector<double> values;
    vector<int> col_indices;
    vector<int> row_ptr;
    int n, m, nnz;

    matCsr() : n(0), m(0), nnz(0) {}
    matCsr(int n0, int m0) : n(n0), m(m0), nnz(0) {
        row_ptr.resize(n0 + 1, 0);
    }

    void append(int n0, int m0, double val) {
        if (abs(val) < 1e-9) return;
        values.push_back(val);
        col_indices.push_back(m0);
        row_ptr[n0 + 1]++;
        nnz++;
    }

    void finalize() {
        for (int i = 0; i < n; ++i) {
            row_ptr[i + 1] += row_ptr[i];
        }
    }

    void createMat(int n0, int m0) {
        n = n0;
        m = m0;
        values.clear();
        col_indices.clear();
        row_ptr.resize(n0 + 1, 0);
        nnz = 0;
    }
};

// Dense matrix class
class mat {
public:
    int n, m;
    vector<double> v; // Change to a single vector to ensure contiguous memory allocation

    mat() {
        m = 1;
        n = 1;
        createMat(1, 1);
    }

    mat(int n0, int m0) {
        m = m0;
        n = n0;
        createMat(n0, m0);
    }

    void createMat(int n0, int m0) {
        n = n0;
        m = m0;
        v.resize(n * m, 0.0); // Ensure contiguous memory allocation
    }

    double& at(int i, int j) {
        return v[i * m + j];
    }

    const double& at(int i, int j) const {
        return v[i * m + j];
    }

    // other member functions ...
};

__global__ void csrMatMultKernel(int n, int m, int k, double* d_values, int* d_col_indices, int* d_row_ptr, double* d_B, double* d_C, int startRow) {
    int row = blockIdx.x * blockDim.x + threadIdx.x + startRow;
    if (row < n) {
        for (int j = 0; j < k; ++j) {
            double sum = 0.0;
            for (int idx = d_row_ptr[row]; idx < d_row_ptr[row + 1]; ++idx) {
                int col = d_col_indices[idx];
                if (col < m) { // Ensure column index is within bounds
                    sum += d_values[idx] * d_B[col * k + j];
                }
            }
            d_C[(row - startRow) * k + j] = sum; // Adjust row index for the offset
        }
    }
}

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " code=" << err << " \"" << hipGetErrorString(err) << "\"" << std::endl; \
            exit(1); \
        } \
    } while (0)

void matMultiplyCPU(int n_threads,const matCsr& csr, const mat& B, mat& C, int startRow, int endRow) {
    int k = B.m;
    omp_set_num_threads(n_threads);
#pragma omp parallel for schedule(dynamic)
    for (int row = startRow; row < endRow; row++) {
        for (int j = 0; j < k; j++) {
            double sum = 0.0;
            for (int idx = csr.row_ptr[row]; idx < csr.row_ptr[row + 1]; ++idx) {
                int col = csr.col_indices[idx];
                if (col < csr.m) { // Ensure column index is within bounds
                    sum += csr.values[idx] * B.at(col, j);
                }
            }
            C.at(row, j) = sum;
        }
    }
}

void matMultiply(const matCsr& csr, const mat& B, mat& C, int k,int n_threads=8) {
    int n = csr.n;
    int m = csr.m;
    int numRowsCPU = n / k;
    int numRowsGPU = n - numRowsCPU;

    C.createMat(n, B.m);

    // CPU part
    thread cpuThread(matMultiplyCPU, n_threads, ref(csr), ref(B), ref(C), 0, numRowsCPU);

    // GPU part
    double* d_values, * d_B, * d_C;
    int* d_col_indices, * d_row_ptr;

    CUDA_CHECK(hipMalloc(&d_values, csr.nnz * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_col_indices, csr.nnz * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_row_ptr, (n + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_B, m * B.m * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, numRowsGPU * B.m * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_values, csr.values.data(), csr.nnz * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_col_indices, csr.col_indices.data(), csr.nnz * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_row_ptr, csr.row_ptr.data(), (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B.v.data(), m * B.m * sizeof(double), hipMemcpyHostToDevice));

    int blockSize = 256;
    int gridSize = (numRowsGPU + blockSize - 1) / blockSize;

    csrMatMultKernel << <gridSize, blockSize >> > (n, m, B.m, d_values, d_col_indices, d_row_ptr, d_B, d_C, numRowsCPU);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(C.v.data() + numRowsCPU * B.m, d_C, numRowsGPU * B.m * sizeof(double), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_values));
    CUDA_CHECK(hipFree(d_col_indices));
    CUDA_CHECK(hipFree(d_row_ptr));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    // Wait for the CPU thread to finish
    cpuThread.join();
}

void generateRandomSparseMatrix(matCsr& csr, int n, int m, int nnz) {
    csr.createMat(n, m);
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<> dis_row(0, n - 1);
    uniform_int_distribution<> dis_col(0, m - 1);
    uniform_real_distribution<> dis_val(0, 100);

    for (int i = 0; i < nnz; ++i) {
        int row = dis_row(gen);
        int col = dis_col(gen);
        double val = dis_val(gen);
        csr.append(row, col, val);
    }
    csr.finalize();
}

void generateRandomMatrix(mat& dense, int n, int m) {
    dense.createMat(n, m);
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<> dis(0, 100);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < m; ++j) {
            dense.at(i, j) = dis(gen);
        }
    }
}

int main() {
    int n = 4096;
    // int m = 4096;
    // int nnz = 1000000; // Number of non-zero elements in CSR matrix
    int splitFactor = 128; // k value for CPU/GPU split

    // matCsr csr;
    // mat B(m, n), C;
    for(int n=1024;n>=16;n>>=1)
    {
        matCsr csr;
        mat B(4096, 4096), C;
        // generateRandomSparseMatrix(csr, n, m, nnz);
        // generateRandomMatrix(B, m, n);
        generateRandomSparseMatrix(csr, 4096, 4096, 1ll*4096*4096/n);
        generateRandomMatrix(B, 4096, 4096);
        for (int spl = 2; spl <= 10; spl++)
        {
            auto start = high_resolution_clock::now();
            for (int i = 1; i < 10; i++)  matMultiply(csr, B, C, spl, 8);
            // for (int i = 1; i < 10; i++)  matMultiply(csr, B, C, thr);
            auto end = high_resolution_clock::now();
            auto duration1 = duration_cast<milliseconds>(end - start).count();
            //cout << "Time Elapsed : " << duration1 << " ms with factor = " <<thr << endl;
            cout << duration1 << " " << n << " "<< spl << endl;
        }
    }
    // for (int thr = 2; thr <= 10; thr++)
    // {
    //     auto start = high_resolution_clock::now();
    //     //for (int i = 1; i < 10; i++)  matMultiply(csr, B, C, splitFactor);
    //     for (int i = 1; i < 10; i++)  matMultiply(csr, B, C, thr);
    //     auto end = high_resolution_clock::now();
    //     auto duration1 = duration_cast<milliseconds>(end - start).count();
    //     //cout << "Time Elapsed : " << duration1 << " ms with factor = " <<thr << endl;
    //     cout << duration1 << " " << thr << endl;
    // }
    // for (int i = 0; i < min(n, 5); ++i) {
    //     for (int j = 0; j < min(n, 5); ++j) {
    //         cout << C.at(i, j) << " ";
    //     }
    //     cout << endl;
    // }

    return 0;
}